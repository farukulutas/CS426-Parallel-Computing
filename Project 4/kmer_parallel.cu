#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_REF_LENGTH 1000000
#define MAX_READ_LENGTH 200
#define MAX_READS 20480

__global__ void countKmers(char *ref, char *reads, int *counts, int *readLengths, int refLen, int numReads, int k, int totalKmers) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= totalKmers) return;

    int readIdx = 0, kmerIdx = tid;
    while (kmerIdx >= readLengths[readIdx] - k + 1) {
        kmerIdx -= readLengths[readIdx] - k + 1;
        readIdx++;
    }

    // Dynamically allocate kmer
    char *kmer = new char[k];
    for (int i = 0; i < k; i++) {
        kmer[i] = reads[readIdx * MAX_READ_LENGTH + kmerIdx + i];
    }

    // Count occurrences in reference
    int count = 0;
    for (int i = 0; i <= refLen - k; i++) {
        bool match = true;
        for (int j = 0; j < k; j++) {
            if (ref[i + j] != kmer[j]) {
                match = false;
                break;
            }
        }
        if (match) count++;
    }
    counts[tid] = count;

    // Free dynamically allocated memory
    delete[] kmer;
}

int main(int argc, char *argv[]) {
    if(argc != 5) {
        printf("Wrong arguments usage: ./kmer_serial [REFERENCE_FILE] [READ_FILE] [k] [OUTPUT_FILE]\n" );
        return 1;
    }

    char *refFile = argv[1];
    char *readsFile = argv[2];
    int k = atoi(argv[3]);
    char *outputFile = argv[4];

    // Read reference sequence
    char ref[MAX_REF_LENGTH] = {0};
    FILE *fp = fopen(refFile, "r");
    if (!fp) {
        printf("Error: File not found %s\n", refFile);
        return 1;
    }
    fgets(ref, MAX_REF_LENGTH, fp);
    fclose(fp);
    int refLen = strlen(ref);

    // Read reads
    char reads[MAX_READS * MAX_READ_LENGTH] = {0};
    int readLengths[MAX_READS] = {0};
    fp = fopen(readsFile, "r");
    if (!fp) {
        printf("Error: File not found %s\n", readsFile);
        return 1;
    }
    char read[MAX_READ_LENGTH];
    int numReads = 0;
    while (fgets(read, MAX_READ_LENGTH, fp) && numReads < MAX_READS) {
        int readLen = strlen(read);
        if (read[readLen - 1] == '\n') read[--readLen] = '\0';
        strcpy(&reads[numReads * MAX_READ_LENGTH], read);
        readLengths[numReads] = readLen;
        numReads++;
    }
    fclose(fp);

    // Calculate totalKmers
    int totalKmers = 0;
    for (int i = 0; i < numReads; i++) {
        totalKmers += readLengths[i] > k ? readLengths[i] - k + 1: 0;
    }

    // Allocate memory on GPU
    char *d_ref, *d_reads;
    int *d_counts, *d_readLengths;
    if (hipMalloc((void **)&d_ref, refLen * sizeof(char)) != hipSuccess ||
        hipMalloc((void **)&d_reads, numReads * MAX_READ_LENGTH * sizeof(char)) != hipSuccess ||
        hipMalloc((void **)&d_counts, totalKmers * sizeof(int)) != hipSuccess ||
        hipMalloc((void **)&d_readLengths, numReads * sizeof(int)) != hipSuccess) {
        printf("Error: hipMalloc failed\n");
        return 1;
    }

    // Copy data to GPU
    hipMemcpy(d_ref, ref, refLen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_reads, reads, numReads * MAX_READ_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_readLengths, readLengths, numReads * sizeof(int), hipMemcpyHostToDevice);

    // Run kernel
    int threadsPerBlock = 256;
    int blocks = (totalKmers + threadsPerBlock - 1) / threadsPerBlock;
    countKmers<<<blocks, threadsPerBlock>>>(d_ref, d_reads, d_counts, d_readLengths, refLen, numReads, k, totalKmers);
    hipDeviceSynchronize();

    // Retrieve results
    int *counts = (int *)malloc(totalKmers * sizeof(int));
    hipMemcpy(counts, d_counts, totalKmers * sizeof(int), hipMemcpyDeviceToHost);

    // Write results to file
    fp = fopen(outputFile, "w");
    for (int i = 0, idx = 0; i < numReads; i++) {
        int totalReadCount = 0;
        for (int j = 0; j < readLengths[i] - k + 1; j++, idx++) {
            totalReadCount += counts[idx];
        }
        fprintf(fp, "%d\n", totalReadCount);
    }
    fclose(fp);

    // Free memory
    hipFree(d_ref);
    hipFree(d_reads);
    hipFree(d_counts);
    hipFree(d_readLengths);
    free(counts);

    return 0;
}
